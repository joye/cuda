#include "hip/hip_runtime.h"
#define BLOCK_SIZE 16

__global__ void MatMulKernel(float* A, float* B, float* C, int width)
{
    __shared__ float Ads[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Bds[BLOCK_SIZE][BLOCK_SIZE];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	int row = by * BLOCK_SIZE + ty;
	int col = bx * BLOCK_SIZE + tx;
	
	float PValue = 0;
	
	for(int m = 0; m < width/BLOCK_SIZE; m++)
	{
	   Ads[ty][tx] = A[row * width + m*BLOCK_SIZE + tx];
	   Bds[ty][tx] = B[(m*BLOCK_SIZE+ty) * width + col];
	   __syncthreads();
	   
	   for(int k = 0; k < BLOCK_SIZE; k++)
	   {
	     PValue += Ads[ty][k]*Bds[k][tx];
	   }
	   __syncthreads();
	}
	C[row*width+col] = PValue;
}


void MatMul(const float* A, const float* B, float* C, int width_A, int height_A, int width_B, int width_B)
{

    float* d_A;
	size_t size = sizeof(A);
	hipMalloc(&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	
	size = sizeof(B);
	float* d_B;
	hipMalloc(&d_B, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	
	float* d_C;
	size_t size_C = height_A*width_B*sizeof(float);
	hipMalloc(&d_C, size_C);
	
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(width_B/BLOCK_SIZE, height_A/BLOCK_SIZE);
	
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width_A);
	
	hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
}